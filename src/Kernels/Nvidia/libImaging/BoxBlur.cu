#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>


__global__ void __cuImagingBoxBlur(const unsigned char* Imin, unsigned char* out, const unsigned w, const unsigned h, int n, float radius){

#define    TILE_W  16
#define    TILE_H  16
#define    Rx      radius
#define    Ry      radius
#define    FILTER_W  (Rx * 2 + 1)
#define    FILTER_H  (Ry * 2 + 1)
#define    BLOCK_W   (TILE_W + (2 * Rx))
#define    BLOCK_H   (TILE_H + (2 * Ry))

        const int x = blockIdx.x * TILE_W - Rx; // x Index
        const int y = blockIdx.y * TILE_H - Ry; // y Index
        const int d = y * w + x ;

        __shared__ float sharedMemory[BLOCK_W][BLOCK_H];
        
        if (x < 0 || y < 0 || x >= w || y >= h){
                sharedMemory[threadIdx.x][threadIdx.y] = 0;
                return;
        }

        sharedMemory[threadIdx.x][threadIdx.y] = Imin[d];
        __syncthreads();

        if ((threadIdx.x >= Rx) && (threadIdx.x < (BLOCK_W-Rx)) && (threadIdx.y >= Ry) && (threadIdx.y < (BLOCK_H-Ry))){
            float sum = 0.0f;
            for(int dx = -Rx; dx <= Rx; dx++){
                for(int dy = -Ry; dy <= Ry; dy++){
                    sum += sharedMemory[threadIdx.x + dx][threadIdx.y + dy];
                        }
                }
            out[d] = sum/FILTER_SIZE;
        }


#undef TILE_W
#undef TILE_H
#undef Rx
#undef Ry
#undef FILTER_W
#undef FILTER_H
#undef BLOCK_W
#undef BLOCK_H
}


void ImagingBoxBlur(Imaging* imOut, Imaging* imIn, float radius, int n){

}